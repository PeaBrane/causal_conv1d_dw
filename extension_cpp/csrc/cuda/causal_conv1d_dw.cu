#include "hip/hip_runtime.h"
#include <cmath>

#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

namespace extension_cpp {

#define KERNEL_SIZE 4
#define BLOCK 32

int cdiv(int a, int b) { return (a + b - 1) / b; }

__device__ __inline__ float sigmoid(float x) { return 1 / (1.0f + expf(-x)); }

__device__ __inline__ float silu(float x) { return x / (1.0f + expf(-x)); }

__device__ __inline__ float silu_jacob(float x) {
  float x_sig = sigmoid(x);
  return x_sig * (1 + x * (1 - x_sig));
}

__global__ void causal_dw_conv1d_fwd_kernel(
  float* input, const float* kernel, float* output, int length, int chs
) {
  __shared__ float s_input[BLOCK][BLOCK];
  __shared__ float s_kernel[KERNEL_SIZE][BLOCK];

  constexpr int bl_stride = BLOCK - KERNEL_SIZE;
  const int tid = threadIdx.x;
  const int b_id = blockIdx.z;
  const int start_pos_id = blockIdx.y * bl_stride - KERNEL_SIZE;
  const int ch_id = blockIdx.x * blockDim.x + tid;

  // load input block into SRAM
  for (int l = 0; l < BLOCK; ++l) {
    int pos_id = start_pos_id + l;
    if (pos_id >= 0 && pos_id < length && ch_id < chs) {
      s_input[l][tid] = input[b_id * length * chs + pos_id * chs + ch_id];
    } else { s_input[l][tid] = 0.0f; }
  }

  // load kernel block into SRAM
  #pragma unroll
  for (int k = 0; k < KERNEL_SIZE; ++k) { s_kernel[k][tid] = kernel[k * chs + ch_id]; }

  // compute output
  for (int l = 1; l <= BLOCK - KERNEL_SIZE; ++l) {
    int store_pos_id = start_pos_id + l + KERNEL_SIZE - 1;
    float sum = 0.0f;
    #pragma unroll
    for (int k = 0; k < KERNEL_SIZE; ++k) { 
      sum += s_kernel[k][tid] * s_input[l + k][tid]; 
    }
    if (store_pos_id < length && ch_id < chs) {
      output[b_id * length * chs + store_pos_id * chs + ch_id] = silu(sum);
    }
  }

}

at::Tensor causal_dw_conv1d_fwd_cuda(const at::Tensor& input, const at::Tensor& kernel) {
  at::Tensor output = torch::empty(input.sizes(), input.options());
  float* input_ptr = input.data_ptr<float>();
  const float* kernel_ptr = kernel.data_ptr<float>();
  float* output_ptr = output.data_ptr<float>();
  // const __half* input_ptr = reinterpret_cast<const __half*>(input_contig.data_ptr<at::Half>());
  // const float* kernel_ptr = kernel_contig.data_ptr<float>();
  // __half* output_ptr = reinterpret_cast<__half*>(output.data_ptr<at::Half>());

  int batch = input.size(0);
  int length = input.size(1);
  int chs = input.size(2);

  dim3 gridDim(cdiv(chs, BLOCK), cdiv(length, BLOCK - KERNEL_SIZE), batch);
  dim3 blockDim(BLOCK, 1, 1);

  causal_dw_conv1d_fwd_kernel<<<gridDim, blockDim>>>(input_ptr, kernel_ptr, output_ptr, length, chs);
  return output;
}

__global__ void causal_dw_conv1d_bwd_kernel(
  const float* input, const float* kernel, const float* grad_output, 
  float* grad_input, float* grad_kernel, 
  int length, int chs
) {
  __shared__ float s_input[BLOCK][BLOCK];
  __shared__ float s_output[BLOCK - KERNEL_SIZE][BLOCK];
  __shared__ float s_kernel[KERNEL_SIZE][BLOCK];

  constexpr int bl_stride = BLOCK - 2 * KERNEL_SIZE;  // halos on both sides
  const int tid = threadIdx.x;
  const int b_id = blockIdx.z;
  const int start_pos_id = blockIdx.y * bl_stride;
  const int ch_id = blockIdx.x * blockDim.x + tid;

  // load input block into SRAM
  for (int l = 0; l < BLOCK; ++l) {
    int pos_id = start_pos_id + l - KERNEL_SIZE;
    if (pos_id >= 0 && pos_id < length && ch_id < chs) {
      s_input[l][tid] = input[b_id * length * chs + pos_id * chs + ch_id];
    } else { s_input[l][tid] = 0.0f; }
  }

  // load kernel block into SRAM
  #pragma unroll
  for (int k = 0; k < KERNEL_SIZE; ++k) { s_kernel[k][tid] = kernel[k * chs + ch_id]; }

  // load grad_output block into SRAM
  for (int l = 0; l < BLOCK - KERNEL_SIZE - 1; ++l) {
    int pos_id = start_pos_id + l;
    int load_id = b_id * length * chs + pos_id * chs + ch_id;
    s_output[l][tid] = (pos_id < length && ch_id < chs) ? grad_output[load_id] : 0.0f;
  }

  // recompute output
  for (int l = 1; l < BLOCK - KERNEL_SIZE; ++l) {
    float sum = 0.0f;
    #pragma unroll
    for (int k = 0; k < KERNEL_SIZE; ++k) { sum += s_kernel[k][tid] * (s_input[l + k][tid]); }
    s_output[l-1][tid] *= silu_jacob(sum);
  }

  // load and modify grad_output block into SRAM
  // for (int l = 0; l < BLOCK - KERNEL_SIZE - 1; ++l) {
  //   int pos_id = start_pos_id + l;
  //   if (pos_id < length && ch_id < chs) {
  //     int load_id = b_id * length * chs + pos_id * chs + ch_id;
  //     s_output[l][tid] = grad_output[load_id] * s_output[l][tid];
  //   } else { s_output[l][tid] = 0.0f; }
  // }
  
  // compute grad_input
  for (int l = 0; l < BLOCK - 2 * KERNEL_SIZE; ++l) {
    int store_pos_id = start_pos_id + l;
    float sum = 0.0f;
    #pragma unroll
    for (int k = 0; k < KERNEL_SIZE; ++k) { sum += s_kernel[KERNEL_SIZE - 1 - k][tid] * s_output[l + k][tid]; }
    if (store_pos_id < length && ch_id < chs) {
      grad_input[b_id * length * chs + store_pos_id * chs + ch_id] = sum;
    }
  }

  // compute grad_kernel
  for (int k = 0; k < KERNEL_SIZE; ++k) {
    int store_id = (KERNEL_SIZE - 1 - k) * gridDim.z * gridDim.y * chs + b_id * gridDim.y * chs + blockIdx.y * chs + ch_id;
    float sum = 0.0f;
    for (int l = 0; l < BLOCK - 2 * KERNEL_SIZE; ++l) {
      sum += s_input[l + KERNEL_SIZE][tid] * s_output[l + k][tid];
    }
    grad_kernel[store_id] = sum;
  }
}

void causal_dw_conv1d_bwd_cuda(
  const at::Tensor& input, const at::Tensor& kernel, const at::Tensor& grad_output, 
  at::Tensor& grad_input, at::Tensor& grad_kernel
) {
  const float* input_ptr = input.data_ptr<float>();
  const float* grad_output_ptr = grad_output.data_ptr<float>();
  const float* kernel_ptr = kernel.data_ptr<float>();
  float* grad_input_ptr = grad_input.data_ptr<float>();
  float* grad_kernel_ptr = grad_kernel.data_ptr<float>();

  int batch = grad_output.size(0);
  int length = grad_output.size(1);
  int chs = grad_output.size(2);

  dim3 gridDim(cdiv(chs, BLOCK), cdiv(length, BLOCK - 2 * KERNEL_SIZE), batch);
  dim3 blockDim(BLOCK, 1, 1);

  causal_dw_conv1d_bwd_kernel<<<gridDim, blockDim>>>(
    input_ptr, kernel_ptr, grad_output_ptr, grad_input_ptr, grad_kernel_ptr, length, chs
  );
}

TORCH_LIBRARY_IMPL(extension_cpp, CUDA, m) {
  m.impl("causal_dw_conv1d_fwd", &causal_dw_conv1d_fwd_cuda);
  m.impl("causal_dw_conv1d_bwd", &causal_dw_conv1d_bwd_cuda);
}

}
